#include "hip/hip_runtime.h"
#include "/usr/local/cuda/compute-sanitizer/include/sanitizer_patching.h"

extern "C" __device__
SanitizerPatchResult SANITIZERAPI my_memory_access_callback(
    void* userdata,
    uint64_t pc,
    void* ptr,
    uint32_t accessSize,
    uint32_t flags)
{
    MyDeviceDataStruct *my_data = (MyDeviceDataStruct *)userdata

    if ((flags & SANITIZER_MEMORY_DEVICE_FLAG_WRITE) != 0)
        // log write
    else
        // log read

    return SANITIZER_PATCH_SUCCESS;
}
    